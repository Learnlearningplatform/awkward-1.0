#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>

#include <iostream>
#include <limits>

/**
 * This is a utility funstion for checking CUDA Errors,
 * NOTE: This function was taken from a blog www.beechwood.eu
 * @param err
 * @param file
 * @param line
 */
static void HandleError(hipError_t err,
                        const char *file,
                        int line) {
  if (err != hipSuccess) {
    int aa = 0;
    printf("%s in %s at line %d\n", hipGetErrorString(err),
           file, line);
    scanf("%d", &aa);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR(err) (HandleError( err, __FILE__, __LINE__ ))

int main() {
  int64_t length;
  length = 1000000;
  int64_t size = length * sizeof(int64_t);

  int64_t *array = new int64_t[length];

  auto max_limit = std::numeric_limits<int32_t>::max();
  srand(time(0));

  for (int64_t i = 0; i < length; i++) {
    int32_t val_1 = rand() % max_limit;
    array[i] = val_1;
  }

  int64_t *d_array;
  HANDLE_ERROR(hipMalloc((void **) &d_array, size));
  HANDLE_ERROR(hipMemcpy(d_array, array, size, hipMemcpyHostToDevice));

  // Experiment1: Does `cuPointerGetAttribute()` work for normal GPU and CPU pointers
  hipPointerAttribute_t attr;
  hipPointerAttribute_t att_1;
  HANDLE_ERROR(hipPointerGetAttributes(&attr, (void *) d_array));

  // This returns an error, CUDA 11 promises to mitigate this by telling that host pointers are  cudaMemoryTypeUnregistered instead of throwing an error
  // This is useful, since now we know that if there's an error it's probably because the pointer is in Main Memory
//  HANDLE_ERROR(cudaPointerGetAttributes(&att_1, (void*) array));
  std::cout << attr.device << "\n";
//  std::cout << att_1.type << "\n";

}